#include "hip/hip_runtime.h"
extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_skein.h"
#include "sph/sph_keccak.h"
#include "sph/sph_cubehash.h"
#include "lyra2/Lyra2.h"
}

#include "miner.h"
#include "cuda_helper.h"
extern "C" {
#include "SHA3api_ref.h"
}
extern void blakeKeccak256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash);
extern void blake256_cpu_setBlock_80(int thr_id, uint32_t *pdata);

extern void keccak256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);
extern void keccak256_cpu_init(int thr_id, uint32_t threads);

extern void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);
extern void skein256_cpu_init(int thr_id, uint32_t threads);

extern void skeinCube256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);


extern void lyra2v2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);

extern void lyra2v2_cpu_init(int thr_id, uint64_t* matrix);

extern void bmw256_cpu_init(int thr_id);
extern void bmw256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *resultnonces, uint32_t target);

extern void cubehash256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_hash);

void lyra2v2_hash(void *state, const void *input)
{
	sph_blake256_context      ctx_blake;
	sph_keccak256_context     ctx_keccak;
	sph_skein256_context      ctx_skein;
	sph_bmw256_context        ctx_bmw;
	sph_cubehash256_context   ctx_cube;

	uint32_t hashA[8], hashB[8];

	sph_blake256_init(&ctx_blake);
	sph_blake256(&ctx_blake, input, 80);
	sph_blake256_close(&ctx_blake, hashA);

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256(&ctx_keccak, hashA, 32);
	sph_keccak256_close(&ctx_keccak, hashB);

	sph_cubehash256_init(&ctx_cube);
	sph_cubehash256(&ctx_cube, hashB, 32);
	sph_cubehash256_close(&ctx_cube, hashA);


	LYRA2(hashB, 32, hashA, 32, hashA, 32, 1, 4, 4, LYRA2_NOBUG);

	sph_skein256_init(&ctx_skein);
	sph_skein256(&ctx_skein, hashB, 32);
	sph_skein256_close(&ctx_skein, hashA);

	sph_cubehash256_init(&ctx_cube);
	sph_cubehash256(&ctx_cube, hashA, 32);
	sph_cubehash256_close(&ctx_cube, hashB);

/*
	sph_bmw256_init(&ctx_bmw);
	sph_bmw256(&ctx_bmw, hashB, 32);
	sph_bmw256_close(&ctx_bmw, hashA);
*/
	BMWHash(256, (const BitSequence*)hashB, 256, (BitSequence*)hashA);

	memcpy(state, hashA, 32);
}

int scanhash_lyra2v2(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	uint32_t *hashes_done)
{
	static THREAD uint64_t *d_hash = nullptr;
	static THREAD uint64_t *d_hash2 = nullptr;

	const uint32_t first_nonce = pdata[19];
#ifdef ORG
	uint32_t intensity = 256 * 256 * 8;
#endif

	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, device_map[thr_id]);
#ifdef ORG
	if(strstr(props.name, "Titan"))
	{
		intensity = 256 * 256 * 15;
#if defined _WIN64 || defined _LP64
		intensity = 256 * 256 * 22;
#endif
	}
	else if(strstr(props.name, "1080"))
	{
		intensity = 256 * 256 * 15;
#if defined _WIN64 || defined _LP64
		intensity = 256 * 256 * 22;
#endif
	}
	else if(strstr(props.name, "1070"))
	{
		intensity = 256 * 256 * 15;
#if defined _WIN64 || defined _LP64
		intensity = 256 * 256 * 22;
#endif
	}
	else if(strstr(props.name, "970"))
	{
		intensity = 256 * 256 * 15;
#if defined _WIN64 || defined _LP64
		intensity = 256 * 256 * 22;
#endif
	}
	else if (strstr(props.name, "980"))
	{
		intensity = 256 * 256 * 15;
#if defined _WIN64 || defined _LP64
		intensity = 256 * 256 * 22;
#endif
	}
	else if (strstr(props.name, "750 Ti"))
	{
		intensity = 256 * 256 * 12;
	}
	else if (strstr(props.name, "750"))
	{
		intensity = 256 * 256 * 5;
	}
	else if (strstr(props.name, "960"))
	{
		intensity = 256 * 256 * 8;
	}
	uint32_t throughputmax = device_intensity(device_map[thr_id], __func__, intensity);
#else
	int dev_id = device_map[thr_id % MAX_GPUS];
	hipGetDeviceProperties(&props, device_map[dev_id]);
	double intensity = 0.001 * (double)props.multiProcessorCount * (double)_ConvertSMVer2Cores(props.major, props.minor) *(double)props.clockRate;

	// Base Spec.
	// RTX2080Ti(PL50) : 78.5MH/s (4352 Core, 1220MHz) : 78,500,000[H/s] * 0.1[s] / 4352 / 1220 = 1.478
	// GTX1060         : 30.4MH/s (1280 Core, 1847MHz) : 30,400,000[H/s] * 0.1[s] / 1280 / 1847 = 1.286
	// GTX750          : 7.64MH/s ( 512 Core, 1137MHz) :  7,640,000[H/s] * 0.1[s] /  512 / 1137 = 1.312
	// GT710           : 1.06MH/s ( 192 Core,  954MHz) :  1,060,000[H/s] * 0.1[s] /  192 /  954 = 0.579
	// GT710           :  296kH/s (  48 Core, 1620MHz) :    296,000[H/s] * 0.1[s] /   48 / 1620 = 0.381
	if (device_sm[dev_id] >= 700) intensity *= 1.478;
	else if (device_sm[dev_id] >= 610) intensity *= 1.286;
	else if (device_sm[dev_id] >= 600) intensity *= 1.478;
	else if (device_sm[dev_id] >= 520) intensity *= 1.286;
	else if (device_sm[dev_id] >= 500) intensity *= 1.312;
	else if (device_sm[dev_id] >= 500) intensity *= 0.579;
	else intensity *= 0.381;

	intensity = (double)((uint32_t)(throughput2intensity((uint32_t)intensity) * 4.0)) * 0.25;

	uint32_t throughputmax = (uint32_t)((1.0 + (intensity - (double)((uint32_t)intensity)))*(1UL << (int)intensity));
	throughputmax = device_intensity(dev_id, __func__, throughputmax);
#endif
	uint32_t throughput = min(throughputmax, max_nonce - first_nonce) & 0xfffffe00;

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x004f;

	static THREAD bool init = false;
	if (!init)
	{ 
#ifdef ORG
		if(throughputmax == intensity)
			applog(LOG_INFO, "GPU #%d: using default intensity %.3f", device_map[thr_id], throughput2intensity(throughputmax));
#else
		intensity = throughput2intensity(throughputmax);
		applog(LOG_WARNING, "Using intensity %2.2f (%d threads)", intensity, throughputmax);
#endif
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		CUDA_SAFE_CALL(hipDeviceReset());
		CUDA_SAFE_CALL(hipSetDeviceFlags(cudaschedule));
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
		CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
#if defined WIN32 && !defined _WIN64
		// 2GB limit for hipMalloc
#ifdef ORG
		if(throughputmax > 0x7fffffffULL / (16 * 4 * 4 * sizeof(uint64_t)))
#else
		if(throughputmax > 0x7fffffffULL / (4 * 4 * sizeof(uint64_t)))
#endif
		{
			applog(LOG_ERR, "intensity too high");
			mining_has_stopped[thr_id] = true;
			hipStreamDestroy(gpustream[thr_id]);
			proper_exit(2);
		}
#endif
#ifdef ORG
		CUDA_SAFE_CALL(hipMalloc(&d_hash2, 16ULL  * 4 * 4 * sizeof(uint64_t) * throughputmax));
#else
		CUDA_SAFE_CALL(hipMalloc(&d_hash2, 4ULL * 4 * sizeof(uint64_t) * throughputmax));
#endif
		CUDA_SAFE_CALL(hipMalloc(&d_hash, 8ULL * sizeof(uint32_t) * throughputmax));

		bmw256_cpu_init(thr_id);
		lyra2v2_cpu_init(thr_id, d_hash2);
		mining_has_stopped[thr_id] = false;

		init = true; 
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	blake256_cpu_setBlock_80(thr_id, pdata);

	do {
		uint32_t foundNonce[2] = { 0, 0 };

		blakeKeccak256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash);
//		keccak256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash);
		cubehash256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash);

		lyra2v2_cpu_hash_32(thr_id, throughput, pdata[19], d_hash);

		skein256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash);
		cubehash256_cpu_hash_32(thr_id, throughput,pdata[19], d_hash);
		bmw256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash, foundNonce, ptarget[7]);

		if(stop_mining)
		{
			mining_has_stopped[thr_id] = true; hipStreamDestroy(gpustream[thr_id]); pthread_exit(nullptr);
		}
		if(foundNonce[0] != 0)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8]={0};
			if(opt_verify)
			{
				be32enc(&endiandata[19], foundNonce[0]);
				lyra2v2_hash(vhash64, endiandata);
			}
			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				// check if there was some other ones...
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (foundNonce[1] != 0)
				{
					if(opt_verify)
					{
						be32enc(&endiandata[19], foundNonce[1]);
						lyra2v2_hash(vhash64, endiandata);
					}
					if(vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{
						pdata[21] = foundNonce[1];
						res++;
						if(opt_benchmark)  applog(LOG_INFO, "GPU #%d Found second nonce %08x", thr_id, foundNonce[1]);
					}
					else
					{
						if(vhash64[7] != Htarg) // don't show message if it is equal but fails fulltest
							applog(LOG_WARNING, "GPU #%d: result does not validate on CPU!", device_map[thr_id]);
					}
				}
				pdata[19] = foundNonce[0];
				if (opt_benchmark) applog(LOG_INFO, "GPU #%d Found nonce % 08x", thr_id, foundNonce[0]);
				return res;
			}
			else
			{
				if (vhash64[7] != Htarg) // don't show message if it is equal but fails fulltest
					applog(LOG_WARNING, "GPU #%d: result does not validate on CPU!", device_map[thr_id]);
			}
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce ;
	return 0;
}
